#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <math.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define BOX_SIZE 23000
/* CUDA error checking helper function */
#define CHECK_CUDA_ERROR(call)                                                 \
  {                                                                            \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(err));                                        \
      return -1;                                                               \
    }                                                                          \
  }

enum platform { CPU, GPU };

typedef struct atomdesc {
  double x_pos;
  double y_pos;
  double z_pos;
} atom;

typedef struct hist_entry {
  unsigned long long d_cnt;
} bucket;

typedef struct histogram {
  bucket *arr;
  unsigned int len;
  double resolution;
} histogram;

typedef struct atom_list {
  atom *arr;
  unsigned long long len;
} atoms_data;

/* Helper function to calculate distance between two points */
__host__ __device__ double p2p_distance(atom a1, atom a2) {
  double x1 = a1.x_pos;
  double x2 = a2.x_pos;
  double y1 = a1.y_pos;
  double y2 = a2.y_pos;
  double z1 = a1.z_pos;
  double z2 = a2.z_pos;

  return sqrt((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) +
              (z1 - z2) * (z1 - z2));
}

/* Core SDH algorithm. Mutates the histogram */
int PDH_baseline(atoms_data *atoms, histogram *hist) {
  int i, j;
  double dist;

  for (i = 0; i < atoms->len; i++) {
    for (j = i + 1; j < atoms->len; j++) {
      dist = p2p_distance(atoms->arr[i], atoms->arr[j]);
      int h_pos = (int)(dist / hist->resolution);
      if (h_pos >= hist->len)
        continue;
      hist->arr[h_pos].d_cnt++;
    }
  }

  return 0;
}

/* CUDA PDH kernel */
__global__ void PDH_cuda_kernel(atom *atoms, long long atoms_len, bucket *hist,
                                int hist_len, double resolution) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  // Filter out-of-bound threads and threads that have x >= y
  if (x >= atoms_len || y >= atoms_len || x >= y)
    return;

  // Calculate the distance between the two atoms
  double dist = p2p_distance(atoms[x], atoms[y]);

  // Calculate the histogram position
  int h_pos = (int)(dist / resolution);
  if (h_pos >= hist_len)
    return;

  // Increment the histogram count with atomic operation
  atomicAdd(&hist[h_pos].d_cnt, 1);
}

/* CUDA PDH algorithm. Mutates the histogram */
int PDH_cuda(atoms_data *atoms_gpu, histogram *hist_gpu, int block_size,
             float *diff) {
  // Check if CUDA device is available
  int deviceCount;
  CHECK_CUDA_ERROR(hipGetDeviceCount(&deviceCount));
  if (deviceCount == 0) {
    fprintf(stderr, "No CUDA devices found\n");
    return -1;
  }

  // Define the number of blocks and threads per block
  // // Maximum x, y dimensions of a block are typically 1024 threads
  dim3 block_dim(block_size, block_size);
  dim3 grid_dim((atoms_gpu->len + block_dim.x - 1) / block_dim.x,
                (atoms_gpu->len + block_dim.y - 1) / block_dim.y);

  hipEvent_t start_time, end_time;
  hipEventCreate(&start_time);
  hipEventCreate(&end_time);
  hipEventRecord(start_time, 0);
  // Launch the kernel
  PDH_cuda_kernel<<<grid_dim, block_dim>>>(atoms_gpu->arr, atoms_gpu->len,
                                           hist_gpu->arr, hist_gpu->len,
                                           hist_gpu->resolution);
  hipEventRecord(end_time, 0);
  hipEventSynchronize(end_time);
  hipEventElapsedTime(diff, start_time, end_time);
  hipEventDestroy(start_time);
  hipEventDestroy(end_time);

  // Check for kernel launch errors
  CHECK_CUDA_ERROR(hipGetLastError());

  // Synchronize to ensure kernel completion
  CHECK_CUDA_ERROR(hipDeviceSynchronize());

  return 0;
}

/* Histogram output function */
void display_histogram(histogram *hist) {
  long long total_cnt = 0;

  for (int i = 0; i < hist->len; i++) {
    if (i % 5 == 0)
      printf("\n%02d: ", i);

    printf("%15lld ", hist->arr[i].d_cnt);
    total_cnt += hist->arr[i].d_cnt;

    if (i == hist->len - 1)
      printf("\n T:%lld \n", total_cnt);
    else
      printf("| ");
  }
}

struct timespec calculate_time(const struct timespec *start,
                               const struct timespec *end) {
  struct timespec diff = {.tv_sec = start->tv_sec - end->tv_sec, //
                          .tv_nsec = start->tv_nsec - end->tv_nsec};
  if (diff.tv_nsec < 0) {
    diff.tv_nsec += 1000000000; // nsec/sec
    diff.tv_sec--;
  }
  return diff;
}

/* Timing and histogram filling function. The algorithm mutates the histogram */
int time_and_fill_histogram_cpu(atoms_data *atoms, histogram *hist,
                                int (*algorithm)(atoms_data *, histogram *),
                                struct timespec *diff) {
  struct timespec start_time;
  struct timespec end_time;
  if (algorithm(atoms, hist) != 0) {
    return -1;
  }

  *diff = calculate_time(&start_time, &end_time);
  return 0;
}

int time_and_fill_histogram_gpu(atoms_data *atoms, histogram *hist,
                                int block_size, float *diff,
                                int (*algorithm)(atoms_data *, histogram *,
                                                 int block_size, float *diff)) {
  if (algorithm(atoms, hist, block_size, diff) != 0) {
    return -1;
  }
  return 0;
}

/* Results calculation and display function */
int calculate_and_display_histogram(atoms_data *atoms, histogram *hist,
                                    platform platform, float *time, int count,
                                    ...) {
  switch (platform) {
  case CPU: {
    printf("Running CPU version\n");
    struct timespec time_diff;
    // Do the calculation and get the time
    if (time_and_fill_histogram_cpu(atoms, hist, PDH_baseline, &time_diff) !=
        0) {
      fprintf(stderr, "Error running the algorithm on the CPU\n");
      return -1;
    }

    // Display histogram
    display_histogram(hist);

    *time = (double)(time_diff.tv_sec * 1000 + time_diff.tv_nsec / 1000000.0);
    return 0;
  }
  case GPU: {
    printf("Running GPU version\n");

    va_list args;
    va_start(args, count);
    int block_size = va_arg(args, int);
    va_end(args);

    // Check if CUDA device is available
    int deviceCount;
    if (hipGetDeviceCount(&deviceCount) != hipSuccess || deviceCount == 0) {
      fprintf(stderr, "No CUDA devices available\n");
      return -1;
    }

    // Initialize data on the GPU
    atom *atoms_arr_gpu;
    bucket *hist_arr_gpu;
    CHECK_CUDA_ERROR(hipMalloc(&atoms_arr_gpu, sizeof(atom) * atoms->len));
    CHECK_CUDA_ERROR(hipMalloc(&hist_arr_gpu, sizeof(bucket) * hist->len));

    // Copy data to GPU with error checking
    CHECK_CUDA_ERROR(hipMemcpy(atoms_arr_gpu, atoms->arr,
                                sizeof(atom) * atoms->len,
                                hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(hist_arr_gpu, hist->arr,
                                sizeof(bucket) * hist->len,
                                hipMemcpyHostToDevice));

    // Initialize the atoms and histogram data structures. Thsese will hold the
    // data on the GPU
    atoms_data atoms_gpu = {atoms_arr_gpu, atoms->len};
    histogram hist_gpu = {hist_arr_gpu, hist->len, hist->resolution};

    // Do the calculation and get the time
    if (time_and_fill_histogram_gpu(&atoms_gpu, &hist_gpu, block_size, time,
                                    PDH_cuda) != 0) {
      fprintf(stderr, "Error running the algorithm on the GPU\n");
      hipFree(atoms_arr_gpu);
      hipFree(hist_arr_gpu);
      return -1;
    }

    // Copy the histogram back to the CPU
    CHECK_CUDA_ERROR(hipMemcpy(hist->arr, hist_gpu.arr,
                                sizeof(bucket) * hist->len,
                                hipMemcpyDeviceToHost));

    hipFree(atoms_arr_gpu);
    hipFree(hist_arr_gpu);

    // Display histogram
    display_histogram(hist);

    return 0;
  }
  }

  // Should never reach here
  return -1;
}

/* Atoms data generation function */
atoms_data init_atoms_data(unsigned int count, int box_size) {
  atom *atoms_arr = (atom *)malloc(sizeof(atom) * count);
  if (atoms_arr == NULL) {
    fprintf(stderr, "Error allocating memory for atoms\n");
    exit(1);
  }
  atoms_data atoms = {
      atoms_arr,
      count,
  };

  // Generate random data points
  srand(1); // Fixed seed for reproducibility
  for (int i = 0; i < atoms.len; i++) {
    atoms.arr[i].x_pos = ((double)(rand()) / RAND_MAX) * box_size;
    atoms.arr[i].y_pos = ((double)(rand()) / RAND_MAX) * box_size;
    atoms.arr[i].z_pos = ((double)(rand()) / RAND_MAX) * box_size;
  }

  return atoms;
}

/* Histogram initialization function */
histogram init_histogram(double resolution, int box_size) {
  // The maximum distance between two points in a box is the diagonal
  unsigned int num_buckets =
      (unsigned int)(box_size * sqrt(3) / resolution) + 1;
  // Allocate the histogram array to store the counts. Initialize to zero
  bucket *hist_arr = (bucket *)calloc(num_buckets, sizeof(bucket));
  if (hist_arr == NULL) {
    fprintf(stderr, "Error allocating memory for histogram\n");
    exit(1);
  }
  histogram hist = {
      hist_arr,
      num_buckets,
      resolution,
  };

  return hist;
}

int main(int argc, char **argv) {
  if (argc != 4) {
    printf("Usage: %s {#of_samples} {bucket_width} {block_size}\n", argv[0]);
    return 1;
  }

  unsigned int particle_count = atoi(argv[1]);
  double resolution = atof(argv[2]);
  int block_size = atoi(argv[3]);

  // Generate heap-allocated data
  atoms_data atoms = init_atoms_data(particle_count, BOX_SIZE);
  histogram hist_cpu = init_histogram(resolution, BOX_SIZE);
  histogram hist_gpu = init_histogram(resolution, BOX_SIZE);

  // Run algorithms
  float time_cpu, time_gpu;
  if (calculate_and_display_histogram(&atoms, &hist_cpu, CPU, &time_cpu, 0) !=
      0) {
    printf("Error running CPU version. Exiting\n");
    free(hist_cpu.arr);
    free(atoms.arr);
    free(hist_gpu.arr);
    return 1;
  }
  if (calculate_and_display_histogram(&atoms, &hist_gpu, GPU, &time_gpu, 1,
                                      block_size) != 0) {
    printf("Error running GPU version. Exiting\n");
    free(hist_cpu.arr);
    free(atoms.arr);
    free(hist_gpu.arr);
    return 1;
  }

  // Calculate the diff histogram (stored in hist_cpu)
  for (int i = 0; i < hist_cpu.len; i++) {
    hist_cpu.arr[i].d_cnt -= hist_gpu.arr[i].d_cnt;
  }

  // Display timing results
  printf("CPU time in miliseconds: %f\n", time_cpu);
  printf("GPU time in miliseconds: %f\n", time_gpu);
  printf("Speedup: %f\n", time_cpu / time_gpu);

  // Display the diff histogram
  printf("Diff histogram:\n");
  display_histogram(&hist_cpu);

  free(hist_gpu.arr);
  free(atoms.arr);
  free(hist_cpu.arr);

  return 0;
}
