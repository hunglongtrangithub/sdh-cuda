// write a simple vector addition kernel
// compile with: nvcc -o test_time test_time.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1000000

#define CHECK_CUDA_ERROR(call)                                                 \
  {                                                                            \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(err));                                        \
      return -1;                                                               \
    }                                                                          \
  }

__global__ void add(int *a, int *b, int *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

int main() {
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;
  printf("N = %d\n", N);
  for (int i = 0; i < N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }
  CHECK_CUDA_ERROR(hipMalloc(&dev_a, N * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc(&dev_b, N * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc(&dev_c, N * sizeof(int)));

  CHECK_CUDA_ERROR(
      hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(
      hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

  add<<<N, 1>>>(dev_a, dev_b, dev_c);
  CHECK_CUDA_ERROR(hipGetLastError());
  CHECK_CUDA_ERROR(hipDeviceSynchronize());

  CHECK_CUDA_ERROR(
      hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

  for (int i = 0; i < N; i++) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  CHECK_CUDA_ERROR(hipFree(dev_a));
  CHECK_CUDA_ERROR(hipFree(dev_b));
  CHECK_CUDA_ERROR(hipFree(dev_c));
  printf("done\n");
  return 0;
}
